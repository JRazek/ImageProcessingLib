#include "hip/hip_runtime.h"
//
// Created by user on 17.09.2021.
//

#include "CUDAConvolutionLayer.cuh"
#include "ConvolutionLayer.h"
#include <memory>
#include "../../../CUDA/CUDAUtils.cuh"

namespace cn{
    __device__
    inline dim3 getDataPos(dim3 bitmapSize, int index){
        if(index >= bitmapSize.x * bitmapSize.y * bitmapSize.z)
            printf("zły arg2 :P");
        return dim3(index % bitmapSize.x, (index % bitmapSize.x * bitmapSize.y) / bitmapSize.x, index / (bitmapSize.x * bitmapSize.y));
    }
    __device__
    inline int getDataIndex(dim3 bitmapSize, dim3 pos){
        if(pos.x >= bitmapSize.x) {
            printf("x, %d %d\n", pos.x, bitmapSize.x);
        }
        if(pos.y >= bitmapSize.y) {
            printf("y, %d %d\n", pos.y, bitmapSize.y);
        }
        if(pos.z >= bitmapSize.z) {
            printf("z, %d %d\n", pos.z, bitmapSize.z);
        }
        return pos.z * bitmapSize.x * bitmapSize.y + pos.y * bitmapSize.x + pos.x;
    }
    __global__
    void CUDAConvAutoGrad(double *chainValues, double *kernelValues, dim3 inputDim, dim3 kernelSize){
        u_int index = blockDim.x * threadIdx.x + threadIdx.x;
        if(index >= inputDim.x * inputDim.y * inputDim.z)
            return;
        dim3 inputPos = getDataPos(inputDim, index);


    }
}
cn::Bitmap<double> cn::CUDAConvolutionLayer::CUDARun(cn::ConvolutionLayer &convolutionLayer, const cn::Bitmap<double> &_input) {
    convolutionLayer.output = std::make_unique<Bitmap<double>>(
            CUDAUtils::cudaConvolve(
                convolutionLayer.kernels, _input,
                convolutionLayer.padding.x, convolutionLayer.padding.y,
                convolutionLayer.stride.x, convolutionLayer.stride.y
            )
    );

    return *convolutionLayer.output.get();
}

void cn::CUDAConvolutionLayer::CUDAAutoGrad(cn::ConvolutionLayer &convolutionLayer) {
    u_int inputSize = convolutionLayer.inputSize.multiplyContent();
    double *inputDev, *kernelDev, *chainValuesDev;


    Bitmap<double> paddedInput = Utils::addPadding(*convolutionLayer.getInput().get(), convolutionLayer.padding.x, convolutionLayer.padding.y);

    u_int paddedInputBytes = paddedInput.size().multiplyContent() * sizeof(double);
    u_int combinedKernelsBytes = convolutionLayer.kernels[0].size().multiplyContent() * convolutionLayer.kernelsCount * sizeof(double);

    inputDev = (double *)CUDAUtils::fixedCudaMalloc(paddedInputBytes);
    kernelDev = (double *)CUDAUtils::fixedCudaMalloc(combinedKernelsBytes);
    chainValuesDev = (double *)CUDAUtils::fixedCudaMalloc(paddedInputBytes);


    std::vector<double> kernelsCombinedData(convolutionLayer.kernels[0].size().multiplyContent() * convolutionLayer.kernelsCount);
    for(u_int i = 0; i < convolutionLayer.kernelsCount; i ++){
        Bitmap<double> &kernel = convolutionLayer.kernels[i];
        std::copy(kernel.dataConst(), kernel.dataConst() + kernel.size().multiplyContent(), kernelsCombinedData.begin() + kernel.size().multiplyContent());
    }



    hipMemcpy(inputDev, paddedInput.data(), paddedInputBytes, hipMemcpyHostToDevice);

    //todo
//    CUDAConvAutoGrad<<<inputSize/cn::THREADS_PER_BLOCK+1, cn::THREADS_PER_BLOCK>>>();

    hipMemcpy(convolutionLayer.memoizationTable->data(), chainValuesDev, paddedInputBytes, hipMemcpyDeviceToHost);

    std::fill(convolutionLayer.memoizationStates->data(), convolutionLayer.memoizationStates->data() + paddedInput.size().multiplyContent(), true);

    hipFree(inputDev);
    hipFree(kernelDev);
    hipFree(chainValuesDev);

}
