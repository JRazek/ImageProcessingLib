#include "hip/hip_runtime.h"
//
// Created by user on 21.09.2021.
//

#include "CUDAOutputLayer.cuh"
#include "OutputLayer.h"
#include "../../../CUDA/CUDAUtils.cuh"

namespace cn{
    __global__
    void CUDACalcOutputGradients(double *output, double *target, double *result, dim3 outputDims){
        u_int index = blockDim.x * blockIdx.x + threadIdx.x;
        if(index >= outputDims.x * outputDims.y * outputDims.z)
            return;
        result[index] = output[index] - target[index];
    }
    inline dim3 vec3ToDim3(const cn::Vector3<int> &vec){
        return dim3(static_cast<u_int>(vec.x), static_cast<u_int>(vec.y), static_cast<u_int>(vec.z));
    }
}

void cn::CUDAOutputLayer::CUDAAutoGrad(cn::OutputLayer &outputLayer) {
    double *outputDev, *targetDev, *resultDev;

    outputDev = (double *) CUDAUtils::fixedCudaMalloc(outputLayer.output->size().multiplyContent() * sizeof(double));
    targetDev = (double *) CUDAUtils::fixedCudaMalloc(outputLayer.target->size().multiplyContent() * sizeof(double));
    resultDev = (double *) CUDAUtils::fixedCudaMalloc(outputLayer.target->size().multiplyContent() * sizeof(double));

    u_int threadsCount = outputLayer.output->size().multiplyContent();

    hipMemcpy(outputDev, outputLayer.output->dataConst(), outputLayer.output->size().multiplyContent() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(targetDev, outputLayer.target->dataConst(), outputLayer.target->size().multiplyContent() * sizeof(double), hipMemcpyHostToDevice);

    CUDACalcOutputGradients<<<threadsCount / cn::THREADS_PER_BLOCK + 1, cn::THREADS_PER_BLOCK>>> (outputDev, targetDev, resultDev, vec3ToDim3(outputLayer.outputSize));

    hipMemcpy(outputLayer.memoizationTable->data(), resultDev, outputLayer.target->size().multiplyContent() * sizeof(double), hipMemcpyDeviceToHost);
    std::fill(outputLayer.memoizationStates->data(), outputLayer.memoizationStates->data() + outputLayer.outputSize.multiplyContent(), true);

    hipFree(outputDev);
    hipFree(targetDev);
    hipFree(resultDev);
}
