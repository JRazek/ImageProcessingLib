#include "hip/hip_runtime.h"
//
// Created by user on 12.09.2021.
//

#include "CUDAUtils.cuh"
#include "../Utils/dataStructures/Bitmap.h"

namespace cn {
    __device__
    int getDataIndex(dim3 bitmapSize, int col, int row, int depth){
        if(col < 0 || col >= bitmapSize.x || row < 0 || row >= bitmapSize.y || depth < 0 || depth >= bitmapSize.z)
            printf("zły arg1 :P");
        return depth * bitmapSize.x * bitmapSize.y + row * bitmapSize.x + col;
    }
    __device__
    dim3 getDataPos(dim3 bitmapSize, int index){
        if(index >= bitmapSize.x * bitmapSize.y * bitmapSize.z)
            printf("zły arg2 :P");
        return dim3(index % bitmapSize.x, (index / bitmapSize.x) % bitmapSize.x, index / (bitmapSize.x * bitmapSize.y));
    }
    __device__
    int afterConvolutionSize(int kernelSize, int inputSize, int padding, int stride) {
        return (inputSize + 2 * padding - kernelSize) / stride + 1;
    }
    __global__
    void cudaConvolveKernel(double *data, double *kernel, dim3 outputSize, dim3 dataSize, dim3 kernelSize, int strideX, int strideY,
                            int paddingX, int paddingY) {
        u_int index = blockIdx.x * blockDim.x + threadIdx.x;
        dim3 kernelPos = getDataPos(outputSize, index);

    }
}

cn::Bitmap<double> cn::CUDAUtils::cudaConvolve(const std::vector<cn::Bitmap<double>> &kernels, const cn::Bitmap<double> &input, int paddingX, int paddingY, int strideX, int strideY) {
    double *kernelDev, *dataDev, *resDev;

    Bitmap<double> paddedInput = cn::Utils::addPadding(input, paddingX, paddingY);

    int sX = cn::Utils::afterConvolutionSize(kernels[0].w(), paddedInput.w(), paddingX, strideX);
    int sY = cn::Utils::afterConvolutionSize(kernels[0].h(), paddedInput.h(), paddingY, strideY);

    u_int kerSize = kernels[0].size().multiplyContent() * sizeof(double) * kernels.size();
    u_int dataSize = paddedInput.size().multiplyContent() * sizeof(double);
    u_int resultSize = sX * sY * kernels.size() * sizeof(double);

    kernelDev = (double *) fixedCudaMalloc(kerSize);
    dataDev = (double *) fixedCudaMalloc(dataSize);
    resDev = (double *) fixedCudaMalloc(resultSize);

    for(int i = 0; i < kernels.size(); i ++){
        hipMemcpy(kernelDev + sX * sY * i, kernels[i].dataConst(), kernels[i].size().multiplyContent() * sizeof(double), hipMemcpyHostToDevice);
    }

    hipMemcpy(dataDev, paddedInput.dataConst(), dataSize, hipMemcpyHostToDevice);

    Bitmap<double> result(sX, sY, kernels.size());

    int threadsCount = result.w() * result.h() * kernels.size();

    constexpr int threadsPerBlock = 1024;


    cudaConvolveKernel<<<threadsCount/threadsPerBlock + 1, std::min(threadsCount, threadsPerBlock)>>>(dataDev, kernelDev,
            {static_cast<u_int>(result.w()), static_cast<u_int>(result.h()), static_cast<u_int>(result.d())},
            {static_cast<u_int>(paddedInput.w()), static_cast<u_int>(paddedInput.h()), static_cast<u_int>(paddedInput.d())},
            {static_cast<u_int>(kernels[0].w()), static_cast<u_int>(kernels[0].h()), static_cast<u_int>(kernels[0].d())},
            strideX, strideY, paddingX, paddingY);



    double *hostRes = new double[sX * sY * kernels.size()];
    hipMemcpy(hostRes, resDev, resultSize, hipMemcpyDeviceToHost);

    result.setData(hostRes);

    delete[] hostRes;

    hipFree(kernelDev);
    hipFree(dataDev);
    hipFree(resDev);

    return result;
}



void *cn::CUDAUtils::fixedCudaMalloc(size_t size) {
    void* tmp;
    return hipMalloc(&tmp, size) == hipError_t::hipSuccess ? tmp : nullptr;
}
