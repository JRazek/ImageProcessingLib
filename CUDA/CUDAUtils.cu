//
// Created by user on 12.09.2021.
//

#include "CUDAUtils.cuh"

int test(){
    return 0;
}

cn::Bitmap<double>
cn::CUDAUtils::cudaConvolve(const cn::Bitmap<double> &kernel, const cn::Bitmap<double> &input, int paddingX, int paddingY, int strideX, int strideY) {
    double *kData,*dData, *result;
    u_int kerSize = kernel.size().multiplyContent() * sizeof(double);
    int dataSize = input.size().multiplyContent() * sizeof(double);

    kData = (double *) fixedCudaMalloc(kerSize);
    dData = (double *) fixedCudaMalloc(dataSize);

    hipMemcpy(kData, kernel.data(), kerSize, hipMemcpyHostToDevice);
    hipMemcpy(dData, input.data(), dataSize, hipMemcpyHostToDevice);



    return cn::Bitmap<double>();
}

__host__
void cn::CUDAUtils::cudaConvolveKernel(double *data, double *kernel, dim3 dataSize, dim3 kernelSize, int strideX,
                                       int strideY, int paddingX,
                                       int paddingY) {
    int sX = cn::Utils::afterConvolutionSize(kernelSize.x, dataSize.x, paddingX, strideX);
    int sY = cn::Utils::afterConvolutionSize(kernelSize.y, dataSize.y, paddingY, strideY);
    u_int index = blockIdx.x * blockDim.x + threadIdx.x;
}

void *cn::CUDAUtils::fixedCudaMalloc(size_t size) {
    void* tmp;
    return hipMalloc(&tmp, size) == hipError_t::hipSuccess ? tmp : nullptr;
}

