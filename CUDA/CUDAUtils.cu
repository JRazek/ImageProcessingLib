#include "hip/hip_runtime.h"
//
// Created by user on 12.09.2021.
//

#include "CUDAUtils.cuh"
#include "../Utils/dataStructures/Bitmap.h"

namespace cn {
    __device__
    int getDataIndex(dim3 bitmapSize, dim3 pos){
        if(pos.x >= bitmapSize.x) {
            printf("x, %d %d\n", pos.x, bitmapSize.x);
        }
        if(pos.y >= bitmapSize.y) {
            printf("y, %d %d\n", pos.y, bitmapSize.y);
        }
        if(pos.z >= bitmapSize.z) {
            printf("z, %d %d\n", pos.z, bitmapSize.z);
        }
        //    return depth * _w * _h + row * _w + col;
        return pos.z * bitmapSize.x * bitmapSize.y + pos.y * bitmapSize.x + pos.x;
    }
    __device__
    dim3 getDataPos(dim3 bitmapSize, int index){
        if(index >= bitmapSize.x * bitmapSize.y * bitmapSize.z)
            printf("zły arg2 :P");
        return dim3(index % bitmapSize.x, (index / bitmapSize.x) % bitmapSize.x, index / (bitmapSize.x * bitmapSize.y));
    }
    __device__
    int afterConvolutionSize(int kernelSize, int inputSize, int padding, int stride) {
        return (inputSize + 2 * padding - kernelSize) / stride + 1;
    }
    __global__
    void cudaConvolveKernel(double *input, double *kernel, double *result, int strideX, int strideY, dim3 inputSize, dim3 outputSize, dim3 kernelSize) {

        u_int index = blockIdx.x * blockDim.x + threadIdx.x;
        u_int posXOutput = index % outputSize.x;
        u_int posYOutput = (index % (outputSize.x * outputSize.y)) / outputSize.x;

        u_int kID = index / (outputSize.x * outputSize.y * inputSize.z); //same as posZOutput
//        printf("kID:%d index:%d \n", kID, index);

        u_int kPosX = posXOutput * strideX;
        u_int kPosY = posYOutput * strideY;
        u_int kPosZ = index / (outputSize.x * outputSize.y);

//        printf("x:%d y:%d z:%d kID:%d\n", kPosX, kPosY, kPosZ, kID);
        double *kernelStart = kernel + kID * (kernelSize.x * kernelSize.y * kernelSize.z);

        double sum = 0;
        for(u_int ky = 0; ky < kernelSize.y; ky++){
            for(u_int kx = 0; kx < kernelSize.x; kx++){
//                if(index > 80)
//                    printf("kdataIndex:%d\n", getDataIndex(kernelSize, {kx, ky, kPosZ}));
                sum += kernelStart[getDataIndex(kernelSize, {kx, ky, kPosZ})] * input[getDataIndex(inputSize, {kPosX + kx, kPosY + ky, kPosZ})];
            }
        }
        result[getDataIndex(outputSize, {posXOutput, posYOutput, kID})] += sum;
    }
}

cn::Bitmap<double> cn::CUDAUtils::cudaConvolve(const std::vector<cn::Bitmap<double>> &kernels, const cn::Bitmap<double> &input, int paddingX, int paddingY, int strideX, int strideY) {
    double *kernelDev, *dataDev, *resDev;

    Bitmap<double> paddedInput = cn::Utils::addPadding(input, paddingX, paddingY);

    int sX = cn::Utils::afterConvolutionSize(kernels[0].w(), input.w(), paddingX, strideX);
    int sY = cn::Utils::afterConvolutionSize(kernels[0].h(), input.h(), paddingY, strideY);

    u_int kerSize = kernels[0].size().multiplyContent() * sizeof(double) * kernels.size();
    u_int dataSize = paddedInput.size().multiplyContent() * sizeof(double);
    u_int resultSize = sX * sY * kernels.size() * sizeof(double);

    kernelDev = (double *) fixedCudaMalloc(kerSize);
    dataDev = (double *) fixedCudaMalloc(dataSize);
    resDev = (double *) fixedCudaMalloc(resultSize);
    hipMemset(resDev, 0, resultSize);
    for(int i = 0; i < kernels.size(); i ++){
        hipMemcpy(kernelDev + sX * sY * i, kernels[i].dataConst(), kernels[i].size().multiplyContent() * sizeof(double), hipMemcpyHostToDevice);
    }

    hipMemcpy(dataDev, paddedInput.dataConst(), dataSize, hipMemcpyHostToDevice);

    Bitmap<double> result(sX, sY, kernels.size());

    int threadsCount = result.w() * result.h() * kernels.size() * paddedInput.d();

    constexpr int threadsPerBlock = 1024;


    dim3 inputSize = {static_cast<u_int>(paddedInput.w()), static_cast<u_int>(paddedInput.h()), static_cast<u_int>(paddedInput.d())};
    dim3 outputSize = {static_cast<u_int>(result.w()), static_cast<u_int>(result.h()), static_cast<u_int>(result.d())};
    dim3 kernelSize = {static_cast<u_int>(kernels[0].w()), static_cast<u_int>(kernels[0].h()), static_cast<u_int>(kernels[0].d())};

    cudaConvolveKernel<<<threadsCount/threadsPerBlock + 1, std::min(threadsCount, threadsPerBlock)>>>
    (
        dataDev,
        kernelDev,
        resDev,
        strideX,
        strideY,
        inputSize,
        outputSize,
        kernelSize
    );



    double *hostRes = new double[sX * sY * kernels.size()];
    hipMemcpy(hostRes, resDev, resultSize, hipMemcpyDeviceToHost);

    result.setData(hostRes);

    delete[] hostRes;

    hipFree(kernelDev);
    hipFree(dataDev);
    hipFree(resDev);

    return result;
}



void *cn::CUDAUtils::fixedCudaMalloc(size_t size) {
    void* tmp;
    return hipMalloc(&tmp, size) == hipError_t::hipSuccess ? tmp : nullptr;
}
