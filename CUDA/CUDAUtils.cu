#include "hip/hip_runtime.h"
//
// Created by user on 12.09.2021.
//

#include "CUDAUtils.cuh"
#include "../Utils/dataStructures/Bitmap.h"

namespace cn {
    __device__
    int getDataIndex(dim3 bitmapSize, dim3 pos){
        if(pos.x >= bitmapSize.x || pos.y >= bitmapSize.y || pos.z >= bitmapSize.z)
            printf("zły arg1 :P");
        return pos.z * bitmapSize.x * bitmapSize.y + pos.y * bitmapSize.x + pos.x;
    }
    __device__
    dim3 getDataPos(dim3 bitmapSize, int index){
        if(index >= bitmapSize.x * bitmapSize.y * bitmapSize.z)
            printf("zły arg2 :P");
        return dim3(index % bitmapSize.x, (index / bitmapSize.x) % bitmapSize.x, index / (bitmapSize.x * bitmapSize.y));
    }
    __device__
    int afterConvolutionSize(int kernelSize, int inputSize, int padding, int stride) {
        return (inputSize + 2 * padding - kernelSize) / stride + 1;
    }
    __global__
    void cudaConvolveKernel(double *input, double *kernel, double *result, dim3 inputSize, dim3 outputSize, int strideX,
                            int strideY, dim3 kernelSize) {
        u_int index = blockIdx.x * blockDim.x + threadIdx.x;
        dim3 kernelPosOutput = getDataPos(outputSize, index);
        dim3 kernelPosInput = {kernelPosOutput.x * strideX, kernelPosOutput.y * strideY, kernelPosOutput.z};
        double sum = 0;
        for(u_int kz = 0; kz < kernelSize.z; kz++) {
            for (u_int ky = 0; ky < kernelSize.y; ky++) {
                for (u_int kx = 0; kx < kernelSize.x; kx++) {
                    dim3 dataPos(kernelPosInput.x + kx, kernelPosInput.y + ky, kernelPosInput.z + kz);
                    sum += kernel[getDataIndex(kernelSize, {kx, ky, kz})] * input[getDataIndex(inputSize, dataPos)];
                }
            }
        }
        result[index] = sum;
    }
}

cn::Bitmap<double> cn::CUDAUtils::cudaConvolve(const std::vector<cn::Bitmap<double>> &kernels, const cn::Bitmap<double> &input, int paddingX, int paddingY, int strideX, int strideY) {
    double *kernelDev, *dataDev, *resDev;

    Bitmap<double> paddedInput = cn::Utils::addPadding(input, paddingX, paddingY);

    int sX = cn::Utils::afterConvolutionSize(kernels[0].w(), paddedInput.w(), paddingX, strideX);
    int sY = cn::Utils::afterConvolutionSize(kernels[0].h(), paddedInput.h(), paddingY, strideY);

    u_int kerSize = kernels[0].size().multiplyContent() * sizeof(double) * kernels.size();
    u_int dataSize = paddedInput.size().multiplyContent() * sizeof(double);
    u_int resultSize = sX * sY * kernels.size() * sizeof(double);

    kernelDev = (double *) fixedCudaMalloc(kerSize);
    dataDev = (double *) fixedCudaMalloc(dataSize);
    resDev = (double *) fixedCudaMalloc(resultSize);

    for(int i = 0; i < kernels.size(); i ++){
        hipMemcpy(kernelDev + sX * sY * i, kernels[i].dataConst(), kernels[i].size().multiplyContent() * sizeof(double), hipMemcpyHostToDevice);
    }

    hipMemcpy(dataDev, paddedInput.dataConst(), dataSize, hipMemcpyHostToDevice);

    Bitmap<double> result(sX, sY, kernels.size());

    int threadsCount = result.w() * result.h() * kernels.size();

    constexpr int threadsPerBlock = 1024;


    cudaConvolveKernel<<<threadsCount/threadsPerBlock + 1, std::min(threadsCount, threadsPerBlock)>>>(dataDev, kernelDev, resDev,
          {static_cast<u_int>(paddedInput.w()),
           static_cast<u_int>(paddedInput.h()),
           static_cast<u_int>(paddedInput.d())},
          {static_cast<u_int>(result.w()),
           static_cast<u_int>(result.h()),
           static_cast<u_int>(result.d())},
          strideX, strideY,
          {static_cast<u_int>(kernels[0].w()),
           static_cast<u_int>(kernels[0].h()),
           static_cast<u_int>(kernels[0].d())}
    );



    double *hostRes = new double[sX * sY * kernels.size()];
    hipMemcpy(hostRes, resDev, resultSize, hipMemcpyDeviceToHost);

    result.setData(hostRes);

    delete[] hostRes;

    hipFree(kernelDev);
    hipFree(dataDev);
    hipFree(resDev);

    return result;
}



void *cn::CUDAUtils::fixedCudaMalloc(size_t size) {
    void* tmp;
    return hipMalloc(&tmp, size) == hipError_t::hipSuccess ? tmp : nullptr;
}
